
#include <hip/hip_runtime.h>
#ifdef WITH_CUDA

#include <cmath>

#include "core/context_cuda.h"
#include "core/tensor.h"
#include "utils/cuda_device.h"
#include "utils/op_kernel.h"
#include "utils/math_functions.h"

namespace dragon {

namespace kernel {

template <typename T>
__global__ void _Empty() { }

template<> void Empty<float, CUDAContext>() {
    _Empty<float> << <1, 1 >> >();
    CUDA_POST_KERNEL_CHECK;
}

template<> void Empty<float16, CUDAContext>() {
    _Empty<float> << <1, 1 >> >();
     CUDA_POST_KERNEL_CHECK;
}

/******************** activation.dropout ********************/

template<typename T>
__global__ void _Dropout(const int count, 
                         const uint32_t thresh, 
                         const T scale, 
                         const T* x, 
                         const uint32_t* mask,
                         T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        y[idx] = x[idx] * (mask[idx] > thresh) * scale;
    }
}

template<> void Dropout<float, CUDAContext>(const int count, 
                                            float prob, 
                                            float scale,
                                            const float* x, 
                                            uint32_t* mask,
                                            float* y, 
                                            CUDAContext* context) {
    uint32_t thresh = static_cast<uint32_t>(UINT_MAX * prob);
    math::RandomUniform<uint32_t, CUDAContext>(count, float(0), float(UINT_MAX), mask);
    _Dropout<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                thresh, 
                                                                 scale, 
                                                                     x, 
                                                                  mask,
                                                                    y);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _DropoutGrad(const int count, 
                             const uint32_t thresh, 
                             const T scale,
                             const T* dy, 
                             const uint32_t* mask,
                             T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        dx[idx] = dy[idx] * (mask[idx] > thresh) * scale;
    }
}

template<> void DropoutGrad<float, CUDAContext>(const int count, 
                                                float prob, 
                                                float scale, 
                                                const float* dy, 
                                                const uint32_t* mask,
                                                float* dx) {
    uint32_t thresh = static_cast<uint32_t>(UINT_MAX * prob);
    _DropoutGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                    thresh, 
                                                                     scale, 
                                                                        dy, 
                                                                      mask,
                                                                       dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** activation.relu ********************/

template <typename T>
__global__ void _Relu(const int count, const T* x, const float slope, T* y) {
    CUDA_KERNEL_LOOP(i, count) {
        y[i] = x[i] > 0 ? x[i] : x[i] * slope;
    }
}

template<> void Relu<float, CUDAContext>(const int count, 
                                         const float* x, 
                                         const float slope, 
                                         float* y) {
    _Relu<float> << < GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, x, slope, y);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _ReluHalf(const int count, const half* x, const float slope, half* y) {
    const half kSlope = __float2half(slope);
    const half kZero = __float2half(0.0);
    CUDA_KERNEL_LOOP(i, count) {
#if __CUDA_ARCH__ >= 530
        y[i] = __hgt(x[i], kZero) ? x[i] : __hmul(x[i], kSlope);
#endif
    }
}

template<> void Relu<float16, CUDAContext>(const int count, 
                                           const float16* x, 
                                           const float slope, 
                                           float16* y) {
    _ReluHalf<half> << < GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                       reinterpret_cast<const half*>(x), 
                                                                  slope, 
                                            reinterpret_cast<half*>(y));
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _ReluGrad(const int count, 
                          const T* dy, 
                          const T* y, 
                          const float slope, 
                          T* dx) {
    CUDA_KERNEL_LOOP(i, count){
        dx[i] = dy[i] * ((y[i] > 0) + slope * (y[i] <= 0));
    }
}

template<> void ReluGrad<float, CUDAContext>(const int count, 
                                             const float* dy, 
                                             const float* y, 
                                             const float slope, 
                                             float* dx) {
    _ReluGrad<float> << < GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                      dy, 
                                                                       y, 
                                                                   slope, 
                                                                     dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** activation.sigmoid ********************/

template <typename T>
__device__ T _SigmoidUnit(const T x) { 
    return T(1) / (T(1) + exp(-x)); 
}

template <typename T>
__global__ void _Sigmoid(const int n, const T* x, T* y) {
    CUDA_KERNEL_LOOP(i, n) {
        y[i] = _SigmoidUnit<T>(x[i]);
    }
}

template<> void Sigmoid<float, CUDAContext>(const int count, const float* x, float* y) {
    _Sigmoid<float> << < GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, x, y);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _SigmoidGrad(const int count, const T* dy, const T* y, T* dx) {
    CUDA_KERNEL_LOOP(i, count) {
        dx[i] = dy[i] * y[i] * (1 - y[i]);
    }
}

template<> void SigmoidGrad<float, CUDAContext>(const int count, 
                                                const float* dy, 
                                                const float* y, 
                                                float* dx) {
    _SigmoidGrad<float> << < GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, dy, y, dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** activation.softmax ********************/

template <typename T>
__global__ void _SoftmaxMaxClass(const int outer_dim, 
                                 const int classes,
                                 const int inner_dim, 
                                 const T* x, 
                                 T* scale) {
    CUDA_KERNEL_LOOP(idx, outer_dim * inner_dim) {
        int o_idx = idx / inner_dim;
        int i_idx = idx % inner_dim;
        T max_val = -FLT_MAX;
        for (int c = 0; c < classes; c++)
            max_val = max(x[(o_idx * classes + c) * inner_dim + i_idx], max_val);
        scale[idx] = max_val;
    }
}

template <typename T>
__global__ void _SoftmaxSubtract(const int count, 
                                 const int classes,
                                 const int inner_dim, 
                                 const T* scale, 
                                 T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        int o_idx = idx / inner_dim / classes;
        int i_idx = idx % inner_dim;
        y[idx] -= scale[o_idx * inner_dim + i_idx];
    }
}

template <typename T>
__global__ void _SoftmaxExp(const int count, T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        y[idx] = std::exp(y[idx]);
    }
}

template <typename T>
__global__ void _SoftmaxSumClass(const int outer_dim, 
                                 const int classes,
                                 const int inner_dim, 
                                 const T* y, 
                                 T* scale) {
    CUDA_KERNEL_LOOP(idx, outer_dim * inner_dim) {
        int o_idx = idx / inner_dim;
        int i_idx = idx % inner_dim;
        T sum = 0;
        for (int c = 0; c < classes; c++)
            sum += y[(o_idx * classes + c) * inner_dim + i_idx];
        scale[idx] = sum;
    }
}

template <typename T>
 __global__ void _SoftmaxDiv(const int count, 
                             const int classes, 
                             const int inner_dim,
                             const T* scale, 
                             T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        int o_idx = idx / inner_dim / classes;
        int i_idx = idx % inner_dim;
        y[idx] /= scale[o_idx * inner_dim + i_idx];
    }
}

template<> void Softmax<float, CUDAContext>(const int count, 
                                            const int classes, 
                                            const int outer_dim, 
                                            const int inner_dim,
                                            const float* sum_multiplier, 
                                            const float* x, 
                                            float* scale, 
                                            float* y,
                                            CUDAContext* context) {
    const int num_preds = inner_dim * outer_dim;
    _SoftmaxMaxClass<float> << <GET_BLOCKS(num_preds), CUDA_NUM_THREADS >> >(outer_dim, 
                                                                               classes, 
                                                                             inner_dim, 
                                                                                     x, 
                                                                                scale);
    _SoftmaxSubtract<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                       classes, 
                                                                     inner_dim, 
                                                                         scale, 
                                                                            y);
    _SoftmaxExp<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, y);
    _SoftmaxSumClass<float> << <GET_BLOCKS(num_preds), CUDA_NUM_THREADS >> >(outer_dim, 
                                                                               classes, 
                                                                             inner_dim, 
                                                                                     y, 
                                                                                scale);
    _SoftmaxDiv<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                  classes, 
                                                                inner_dim, 
                                                                    scale, 
                                                                       y);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _SoftmaxDot(const int outer_dim, 
                            const int classes, 
                            const int inner_dim,
                            const T* dy, 
                            const T* y, 
                            T* scale) {
    CUDA_KERNEL_LOOP(idx, outer_dim * inner_dim) {
        int o_idx = idx / inner_dim;
        int i_idx = idx % inner_dim;
        T dot = 0;
        for (int c = 0; c < classes; c++)
            dot += (y[(o_idx * classes + c) * inner_dim + i_idx] * 
                   dy[(o_idx * classes + c) * inner_dim + i_idx]);
        scale[idx] = dot;
    }
}

template<> void SoftmaxGrad<float, CUDAContext>(const int count, 
                                                const int classes, 
                                                const int outer_dim, 
                                                const int inner_dim,
                                                const float* sum_multiplier, 
                                                const float* dy, 
                                                const float* y, 
                                                float* scale, 
                                                float* dx) {
    const int num_preds = inner_dim * outer_dim;
    _SoftmaxDot<float> << <GET_BLOCKS(num_preds), CUDA_NUM_THREADS >> >(outer_dim,
                                                                          classes, 
                                                                        inner_dim, 
                                                                               dy, 
                                                                                y, 
                                                                           scale);
    _SoftmaxSubtract<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                       classes, 
                                                                     inner_dim, 
                                                                         scale, 
                                                                           dx);
    math::Mul<float, CUDAContext>(count, dx, y, dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** activation.tanh ********************/

template <typename T>
__global__ void _Tanh(const int count, const T* x, T* y) {
    CUDA_KERNEL_LOOP(i, count) {
        y[i] = std::tanh(x[i]);
    }
}

template<> void Tanh<float, CUDAContext>(const int count, const float* x, float* y) {
    _Tanh<float> << < GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, x, y);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _TanhGrad(const int count, const T* dy, const T* y, T* dx) {
    CUDA_KERNEL_LOOP(i, count) {
        dx[i] = dy[i] * (1 - y[i] * y[i]);
    }
}

template<> void TanhGrad<float, CUDAContext>(const int count, 
                                             const float* dy, 
                                             const float* y, 
                                             float* dx) {
    _TanhGrad<float> << < GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, dy, y, dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** arithmetic.bias_add ********************/

template <typename T>
__global__ void _BiasAddNCHW(const int count, 
                             const int dim, 
                             const int inner_dim,
                             const T* bias, 
                             T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int bias_idx = (idx / inner_dim) % dim;
        y[idx] += bias[bias_idx];
    }
}

template<> void BiasAdd<float, CUDAContext>(const int count, 
                                            const int outer_dim, 
                                            const int dim, 
                                            const int inner_dim,
                                            const string& format, 
                                            const float* bias, 
                                            const float* bias_multiplier, 
                                            float* y) {
    if (format == "NCHW") {
        _BiasAddNCHW<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                           dim, 
                                                                     inner_dim, 
                                                                          bias, 
                                                                            y);
    } else { NOT_IMPLEMENTED; }
}

/******************** arithmetic.clip ********************/

template <typename T>
__global__ void _Clip(const int count, 
                      const T low, 
                      const T high, 
                      const T* x,
                      T* mask,
                      T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        mask[idx] = 1.0;
        if (x[idx] > high || x[idx] < low) mask[idx] = 0.0;
        y[idx] = x[idx] > high ? high : x[idx];
        y[idx] = x[idx] < low ? low : x[idx];
    }
}

template <> void Clip<float, CUDAContext>(const int count,
                                          const float low,
                                          const float high,
                                          const float* x,
                                          float* mask,
                                          float* y) {
    _Clip<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                low, 
                                                               high, 
                                                                  x,
                                                               mask,
                                                                 y);
}

/******************** arithmetic.scale ********************/

template <typename T>
__global__ void _ScaleWithoutBias(const int n, 
                                  const T* x, 
                                  const T* scale,
                                  const int scale_dim, 
                                  const int inner_dim, 
                                  T* y) {
    CUDA_KERNEL_LOOP(idx, n) {
        const int scale_idx = (idx / inner_dim) % scale_dim;
         y[idx] = x[idx] * scale[scale_idx];
    }
}

template <typename T>
__global__ void _ScaleWithBias(const int n, 
                               const T* x, 
                               const T* scale, 
                               const T* bias, 
                               const int scale_dim, 
                               const int inner_dim, 
                               T* y) {
    CUDA_KERNEL_LOOP(idx, n) {
        const int scale_idx = (idx / inner_dim) % scale_dim;
        y[idx] = x[idx] * scale[scale_idx] + bias[scale_idx];
    }
}

template<> void Scale<float, CUDAContext>(const int axis, 
                                          Tensor* x, 
                                          Tensor* gamma,
                                          Tensor* beta, 
                                          Tensor* BMul, 
                                          Tensor* y) {
    const int count = x->count();
    const int inner_dim = x->count(axis + gamma->ndim());
    const int scale_dim = gamma->count();
    auto* Xdata = x->data<float, CUDAContext>();
    auto* Ydata = y->mutable_data<float, CUDAContext>();
    auto* Sdata = gamma->data<float, CUDAContext>();
    auto* Bdata = beta != nullptr ? 
                          beta->data<float, CUDAContext>() : 
                          nullptr;
    if (Bdata != nullptr)
        _ScaleWithBias<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                           Xdata, 
                                                                           Sdata, 
                                                                           Bdata, 
                                                                       scale_dim, 
                                                                       inner_dim, 
                                                                          Ydata);
    else _ScaleWithoutBias<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                               Xdata, 
                                                                               Sdata, 
                                                                           scale_dim, 
                                                                           inner_dim, 
                                                                              Ydata);
}

template <typename T>
__global__ void _ScaleWithoutBiasHalf(const int n, 
                                      const half* x, 
                                      const half* scale,
                                      const int scale_dim, 
                                      const int inner_dim, 
                                      half* y) {
    CUDA_KERNEL_LOOP(idx, n) {
#if __CUDA_ARCH__ >= 530
        const int scale_idx = (idx / inner_dim) % scale_dim;
        y[idx] = __hmul(x[idx], scale[scale_idx]);
#endif
    }
}

template <typename T>
__global__ void _ScaleWithBiasHalf(const int n, 
                                   const half* x, 
                                   const half* scale, 
                                   const half* bias, 
                                   const int scale_dim, 
                                   const int inner_dim, 
                                   half* y) {
    CUDA_KERNEL_LOOP(idx, n) {
#if __CUDA_ARCH__ >= 530
        const int scale_idx = (idx / inner_dim) % scale_dim;
        y[idx] = __hadd(__hmul(x[idx], scale[scale_idx]), bias[scale_idx]);
#endif
    }
}

template<> void Scale<float16, CUDAContext>(const int axis, 
                                            Tensor* x, 
                                            Tensor* gamma,
                                            Tensor* beta, 
                                            Tensor* BMul, 
                                            Tensor* y) {
    const int count = x->count();
    const int inner_dim = x->count(axis + gamma->ndim());
    const int scale_dim = gamma->count();
    auto* Xdata = x->data<float16, CUDAContext>();
    auto* Ydata = y->mutable_data<float16, CUDAContext>();
    auto* Sdata = gamma->data<float16, CUDAContext>();
    auto* Bdata = beta != nullptr ? 
                          beta->data<float16, CUDAContext>() :
                          nullptr;
    if (Bdata != nullptr)
        _ScaleWithBiasHalf<half> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                               reinterpret_cast<const half*>(Xdata),
                                               reinterpret_cast<const half*>(Sdata),
                                               reinterpret_cast<const half*>(Bdata),
                                                                          scale_dim, 
                                                                          inner_dim, 
                                                    reinterpret_cast<half*>(Ydata));
    else _ScaleWithoutBiasHalf<half> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                   reinterpret_cast<const half*>(Xdata),
                                                   reinterpret_cast<const half*>(Sdata),
                                                                              scale_dim, 
                                                                              inner_dim, 
                                                        reinterpret_cast<half*>(Ydata));
}

template <> void ScaleGrad<float, CUDAContext>(const int axis, 
                                               Tensor* dy, 
                                               Tensor* gamma, 
                                               Tensor* dx) {
    const int count = dx->count();
    const int inner_dim = dx->count(axis + gamma->ndim());
    const int scale_dim = gamma->count();
    auto* dYdata = dy->data<float, CUDAContext>();
    auto* dXdata = dx->mutable_data<float, CUDAContext>();
    auto* Sdata = gamma->data<float, CUDAContext>();
    _ScaleWithoutBias<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                         dYdata, 
                                                                          Sdata, 
                                                                      scale_dim, 
                                                                      inner_dim, 
                                                                        dXdata);
}

/******************** common.argmax ********************/

template <typename T>
__global__ void _Argmax(const int count, 
                        const int axis_dim, 
                        const int inner_dim, 
                        const T* x, 
                        T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        T max_val = -FLT_MAX;
        int max_idx = -1;
        for (int j = 0; j < axis_dim; ++j) {
            const T val = x[(idx / inner_dim * axis_dim + j) 
                                * inner_dim + idx % inner_dim];
            if (val > max_val) {
                max_val = val;
                max_idx = j;
            }
        }
        y[idx] = max_idx;
    }
}

template<> void Argmax<float, CUDAContext>(const int count, 
                                           const int axis_dim, 
                                           const int inner_dim, 
                                           const int top_k, 
                                           const float* x, 
                                           float* y) {
    CHECK_EQ(top_k, 1) << "top_k > 1 is not implemented with CUDA";
    _Argmax<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                             axis_dim, 
                                                            inner_dim, 
                                                                    x, 
                                                                   y);
    CUDA_POST_KERNEL_CHECK;
}

/******************** common.at ********************/

template <typename T>
__global__ void _CanonicalAxis(const int count, const int dim, T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        if (y[idx] < 0) y[idx] += dim;
    }
}

template <> void CanonicalAxis<float, CUDAContext>(const int count, const int dim, float* y) {
    _CanonicalAxis<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, dim, y);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _At(const int count, 
                    const int outer_dim, 
                    const int inner_dim,
                    const int x_slice_dim, 
                    const int y_slice_dim, 
                    const T* indices, 
                    const T* x, 
                    T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int outer_idx = idx / inner_dim / y_slice_dim;
        const int slice_idx = idx % inner_dim;
        const int y_idx_offset = (idx / inner_dim) % y_slice_dim;
        const int x_idx_offset = indices[y_idx_offset];
        const int x_idx = (outer_idx * x_slice_dim + x_idx_offset)
                                     * inner_dim + slice_idx;
        y[idx] = x[x_idx];
    }
}

template <> void At<float, CUDAContext>(const int count, 
                                        const int outer_dim, 
                                        const int inner_dim,
                                        const int x_slice_dim, 
                                        const int y_slice_dim, 
                                        const float* indices,
                                        const float* x, 
                                        float* y, 
                                        CUDAContext* context) {
    _At<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                        outer_dim, 
                                                        inner_dim, 
                                                      x_slice_dim, 
                                                      y_slice_dim,
                                                          indices, 
                                                                x, 
                                                               y);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _AtGrad(const int count, 
                        const int outer_dim, 
                        const int inner_dim,
                        const int x_slice_dim, 
                        const int y_slice_dim, 
                        const T* indices, 
                        const T* dy, 
                        T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int outer_idx = idx / inner_dim / y_slice_dim;
        const int slice_idx = idx % inner_dim;
        const int y_idx_offset = (idx / inner_dim) % y_slice_dim;
        const int x_idx_offset = indices[y_idx_offset];
        const int x_idx = (outer_idx * x_slice_dim + x_idx_offset)
                                     * inner_dim + slice_idx;
        atomicAdd(dx + x_idx, dy[idx]);
    }
}

template <> void AtGrad<float, CUDAContext>(const int count, 
                                            const int outer_dim, 
                                            const int inner_dim,
                                            const int x_slice_dim, 
                                            const int y_slice_dim, 
                                            const float* indices,
                                            const float* dy, 
                                            float* dx, 
                                            CUDAContext* context) {
    _AtGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                            outer_dim, 
                                                            inner_dim, 
                                                          x_slice_dim, 
                                                          y_slice_dim,
                                                              indices, 
                                                                   dy, 
                                                                  dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** common.concat ********************/

template <typename T>
__global__ void _Concat(const int count, 
                        const int outer_dim, 
                        const int inner_dim,
                        const int x_concat_dim, 
                        const int y_concat_dim, 
                        const int concat_offset, 
                        const T* x, 
                        T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int tmp = x_concat_dim * inner_dim;
        const int outer_idx = idx / tmp;
        const int concat_idx = idx % tmp;
        const int y_idx = (outer_idx * y_concat_dim + concat_offset) 
                                     * inner_dim + concat_idx;
        y[y_idx] = x[idx];
    }
}

template <> void Concat<float, CUDAContext>(const int count, 
                                            const int outer_dim, 
                                            const int inner_dim,
                                            const int x_concat_dim, 
                                            const int y_concat_dim, 
                                            const int concat_offset,
                                            const float* x, 
                                            float* y, 
                                            CUDAContext* context) {
    _Concat<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                            outer_dim, 
                                                            inner_dim, 
                                                         x_concat_dim, 
                                                         y_concat_dim,
                                                        concat_offset, 
                                                                    x, 
                                                                   y);
    CUDA_POST_KERNEL_CHECK;
}

template <> void Concat<float16, CUDAContext>(const int count, 
                                              const int outer_dim, 
                                              const int inner_dim,
                                              const int x_concat_dim, 
                                              const int y_concat_dim, 
                                              const int concat_offset,
                                              const float16* x, 
                                              float16* y, 
                                              CUDAContext* context) {
    _Concat<half> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                           outer_dim, 
                                                           inner_dim, 
                                                        x_concat_dim, 
                                                        y_concat_dim,
                                                       concat_offset, 
                                    reinterpret_cast<const half*>(x),
                                         reinterpret_cast<half*>(y));
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _ConcatGrad(const int count, 
                            const int outer_dim, 
                            const int inner_dim,
                            const int x_concat_dim, 
                            const int y_concat_dim, 
                            const int concat_offset, 
                            const T* dy, 
                            T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int tmp = x_concat_dim * inner_dim;
        const int outer_idx = idx / tmp;
        const int concat_idx = idx % tmp;
        const int y_idx = (outer_idx * y_concat_dim + concat_offset)
                                     * inner_dim + concat_idx;
        dx[idx] = dy[y_idx];
    }
}

template <> void ConcatGrad<float, CUDAContext>(const int count, 
                                                const int outer_dim, 
                                                const int inner_dim,
                                                const int x_concat_dim, 
                                                const int y_concat_dim, 
                                                const int concat_offset,
                                                const float* dy, 
                                                float* dx, 
                                                CUDAContext* context) {
    _ConcatGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                outer_dim, 
                                                                inner_dim, 
                                                             x_concat_dim, 
                                                             y_concat_dim,
                                                            concat_offset, 
                                                                       dy, 
                                                                      dx);
    CUDA_POST_KERNEL_CHECK;
}

template <> void ConcatGrad<float16, CUDAContext>(const int count, 
                                                  const int outer_dim, 
                                                  const int inner_dim,
                                                  const int x_concat_dim, 
                                                  const int y_concat_dim, 
                                                  const int concat_offset,
                                                  const float16* dy, 
                                                  float16* dx, 
                                                  CUDAContext* context) {
    _ConcatGrad<half> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                               outer_dim, 
                                                               inner_dim, 
                                                            x_concat_dim, 
                                                            y_concat_dim,
                                                           concat_offset, 
                                       reinterpret_cast<const half*>(dy),
                                            reinterpret_cast<half*>(dx));
    CUDA_POST_KERNEL_CHECK;
}

/******************** common.crop ********************/

template<typename T>
__global__ void _Crop2D(const int count, 
                        const int x_w_dim, 
                        const int y_w_dim, 
                        const int x_h_offset,
                        const int x_w_offset,
                        const T* x, 
                        T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int y_w = idx % y_w_dim;
        const int y_h = (idx / y_w_dim);
        y[idx] = x[(y_h + x_h_offset) * x_w_dim + x_w_offset + y_w];
    }
}

template<> void Crop2D<float, CUDAContext>(vector<TIndex> idxs,
                                           const vector<TIndex>& offsets,
                                           const int cur_dim,
                                           Tensor* x,
                                           Tensor* y,
                                           CUDAContext* context) {
    TIndex inner_dim = 1;
    for (int i = 0; i < 2; i++) inner_dim *= y->dim(cur_dim + i);
    TIndex x_w_dim = x->dim(cur_dim + 1), y_w_dim = y->dim(cur_dim + 1);
    TIndex x_h_offset = offsets[cur_dim], x_w_offset = offsets[cur_dim + 1];

    auto* Xdata = x->data<float, CUDAContext>();
    auto* Ydata = y->mutable_data<float, CUDAContext>();
    Xdata += x->offset(idxs);
    Ydata += y->offset(idxs);

    _Crop2D<float> << <GET_BLOCKS(inner_dim), CUDA_NUM_THREADS >> >(inner_dim,
                                                                      x_w_dim,
                                                                      y_w_dim,
                                                                   x_h_offset,
                                                                   x_w_offset,
                                                                        Xdata,
                                                                       Ydata);
    CUDA_POST_KERNEL_CHECK;
}

template<typename T>
__global__ void _Crop2DGrad(const int count, 
                            const int x_w_dim, 
                            const int y_w_dim, 
                            const int x_h_offset,
                            const int x_w_offset,
                            const T* dy, 
                            T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int y_w = idx % y_w_dim;
        const int y_h = (idx / y_w_dim);
        dx[(y_h + x_h_offset) * x_w_dim + x_w_offset + y_w] = dy[idx];
    }
}

template<> void Crop2DGrad<float, CUDAContext>(vector<TIndex> idxs,
                                               const vector<TIndex>& offsets,
                                               const int cur_dim,
                                               Tensor* dy,
                                               Tensor* dx,
                                               CUDAContext* context) {
    TIndex inner_dim = 1;
    for (int i = 0; i < 2; i++) inner_dim *= dy->dim(cur_dim + i);
    TIndex x_w_dim = dx->dim(cur_dim + 1), y_w_dim = dy->dim(cur_dim + 1);
    TIndex x_h_offset = offsets[cur_dim], x_w_offset = offsets[cur_dim + 1];

    auto* dYdata = dy->data<float, CUDAContext>();
    auto* dXdata = dx->mutable_data<float, CUDAContext>();
    dYdata += dy->offset(idxs);
    dXdata += dx->offset(idxs);

    _Crop2DGrad<float> << <GET_BLOCKS(inner_dim), CUDA_NUM_THREADS >> >(inner_dim,
                                                                          x_w_dim,
                                                                          y_w_dim,
                                                                       x_h_offset,
                                                                       x_w_offset,
                                                                           dYdata,
                                                                          dXdata);
    CUDA_POST_KERNEL_CHECK;
}

/******************** common.reduce ********************/

template <typename T>
__global__ void _Sum(const int count, 
                     const int axis_dim,
                     const int inner_dim, 
                     const T* x, 
                     float* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        T sum_val = 0.0;
        for (int j = 0; j < axis_dim; j++)
            sum_val += x[(idx / inner_dim * axis_dim + j) 
                          * inner_dim + idx % inner_dim];
        y[idx] = sum_val;
   }
}

template<> void Sum<float, CUDAContext>(
        const int count, const int axis_dim,
        const int inner_dim, const float* x, float* y){
    _Sum<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                          axis_dim, 
                                                         inner_dim, 
                                                                 x, 
                                                                y);
     CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _SumGrad(const int count, 
                         const int axis_dim,
                         const int inner_dim, 
                         const T coeff, 
                         const T* dy, 
                         float* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        for (int j = 0; j < axis_dim; j++)
            dx[(idx / inner_dim * axis_dim + j) 
                    * inner_dim + idx % inner_dim] = dy[idx] * coeff;
    }
}

template<> void SumGrad<float, CUDAContext>(const int count, 
                                            const int axis_dim, 
                                            const int inner_dim, 
                                            const float coeff, 
                                            const float* dy, 
                                            float* dx) {
    _SumGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                              axis_dim, 
                                                             inner_dim,
                                                                 coeff, 
                                                                    dy, 
                                                                   dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** common.slice ********************/

template <typename T>
    __global__ void _Slice(const int count, const int outer_dim, const int inner_dim,
        const int x_slice_dim, const int y_slice_dim, const int slice_offset, const T* x, T* y){
        CUDA_KERNEL_LOOP(idx, count) {
            const int tmp = y_slice_dim * inner_dim;
            const int outer_idx = idx / tmp;
            const int slice_idx = idx % tmp;
            const int x_idx = (outer_idx * x_slice_dim + slice_offset)
                * inner_dim + slice_idx;
            y[idx] = x[x_idx];
        }
}

template <> void Slice<float, CUDAContext>(const int count, 
                                           const int outer_dim, 
                                           const int inner_dim,
                                           const int x_slice_dim, 
                                           const int y_slice_dim, 
                                           const int slice_offset,
                                           const float* x, 
                                           float* y, 
                                           CUDAContext* context) {
    _Slice<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                           outer_dim, 
                                                           inner_dim, 
                                                         x_slice_dim, 
                                                         y_slice_dim, 
                                                        slice_offset, 
                                                                   x, 
                                                                  y);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _SliceGrad(const int count, 
                           const int outer_dim, 
                           const int inner_dim,
                           const int x_slice_dim, 
                           const int y_slice_dim, 
                           const int slice_offset, 
                           const T* dy, 
                           T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int tmp = y_slice_dim * inner_dim;
        const int outer_idx = idx / tmp;
        const int slice_idx = idx % tmp;
        const int x_idx = (outer_idx * x_slice_dim + slice_offset)
                                     * inner_dim + slice_idx;
        dx[x_idx] = dy[idx];
    }
}

template <> void SliceGrad<float, CUDAContext>(const int count, 
                                               const int outer_dim, 
                                               const int inner_dim,
                                               const int x_slice_dim, 
                                               const int y_slice_dim, 
                                               const int slice_offset,
                                               const float* dy, 
                                               float* dx, 
                                               CUDAContext* context) {
    _SliceGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                               outer_dim, 
                                                               inner_dim, 
                                                             x_slice_dim, 
                                                             y_slice_dim,
                                                            slice_offset, 
                                                                      dy, 
                                                                     dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** common.tile ********************/

template <typename T>
__global__ void _Tile(const int count, 
                      const int inner_dim, 
                      const int multiple, 
                      const int dim, 
                      const T* x, 
                      T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int d = idx % inner_dim;
        const int b = (idx / inner_dim / multiple) % dim;
        const int n = idx / inner_dim / multiple / dim;
        const int x_idx = (n * dim + b) * inner_dim + d;
        y[idx] = x[x_idx];
    }
}

template <> void Tile<float, CUDAContext>(const int count, 
                                          const int outer_dim, 
                                          const int inner_dim,
                                          const int dim,
                                          const int multiple, 
                                          const float* x, 
                                          float* y, 
                                          CUDAContext* context) {
    _Tile<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                          inner_dim, 
                                                           multiple, 
                                                                dim, 
                                                                  x, 
                                                                 y);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _TileGrad(const int count, 
                          const int inner_dim,
                          const int multiple, 
                          const int dim, 
                          const T* dy, 
                          T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int d = idx % inner_dim;
        const int b = (idx / inner_dim) % dim;
        const int n = idx / inner_dim / dim;
        int y_idx = (n * multiple * dim + b) * inner_dim + d;
        dx[idx] = 0;
        for (int t = 0; t < multiple; t++) {
            dx[idx] += dy[y_idx];
            dy += dim * inner_dim;
        }
    }
}

template <> void TileGrad<float, CUDAContext>(const int count, 
                                              const int outer_dim, 
                                              const int inner_dim, 
                                              const int dim,
                                              const int multiple, 
                                              const float* dy, 
                                              float* dx, 
                                              CUDAContext* context) {
    _TileGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                        inner_dim / dim, 
                                                               multiple, 
                                                                    dim, 
                                                                     dy, 
                                                                    dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** common.transpose ********************/

template <typename T>
__global__ void _Transpose(const int count, 
                           const int ndim, 
                           const int* order, 
                           const int* old_steps, 
                           const int* new_steps, 
                           const T* x, 
                           T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
       int x_idx = 0, y_idx = idx;
       for (int j = 0; j < ndim; ++j) {
           int k = order[j];
           x_idx += (y_idx / new_steps[j]) * old_steps[k];
           y_idx %= new_steps[j];
       }
       y[idx] = x[x_idx];
   }
}

template <> void Transpose<float, CUDAContext>(const int count, 
                                               const int ndim, 
                                               const int* order, 
                                               const int* old_steps,
                                               const int* new_steps, 
                                               const float* x, 
                                               float* y) {
    _Transpose<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                    ndim, 
                                                                   order, 
                                                               old_steps, 
                                                               new_steps, 
                                                                       x, 
                                                                      y);
    CUDA_POST_KERNEL_CHECK;
}

template <> void Transpose<float16, CUDAContext>(const int count, 
                                                 const int ndim, 
                                                 const int* order, 
                                                 const int* old_steps,
                                                 const int* new_steps, 
                                                 const float16* x, 
                                                 float16* y) {
    _Transpose<half> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                   ndim, 
                                                                  order, 
                                                              old_steps, 
                                                              new_steps, 
                                       reinterpret_cast<const half*>(x),
                                            reinterpret_cast<half*>(y));
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _TransposeGrad(const int count, 
                               const int ndim, 
                               const int* order,
                               const int* old_steps, 
                               const int* new_steps,
                               const T* dy, 
                               T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        int x_idx = 0, y_idx = idx;
        for (int j = 0; j < ndim; ++j) {
            int k = order[j];
            x_idx += (y_idx / new_steps[j]) * old_steps[k];
            y_idx %= new_steps[j];
        }
        dx[x_idx] = dy[idx];
    }
}

template <> void TransposeGrad<float, CUDAContext>(const int count, 
                                                   const int ndim,
                                                   const int* order, 
                                                   const int* old_steps,
                                                   const int* new_steps, 
                                                   const float* dy, 
                                                   float* dx) {
    _TransposeGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                        ndim, 
                                                                       order, 
                                                                   old_steps, 
                                                                   new_steps, 
                                                                          dy, 
                                                                         dx);
    CUDA_POST_KERNEL_CHECK;
}

template <> void TransposeGrad<float16, CUDAContext>(const int count, 
                                                     const int ndim,
                                                     const int* order, 
                                                     const int* old_steps,
                                                     const int* new_steps, 
                                                     const float16* dy, 
                                                     float16* dx) {
    _TransposeGrad<half> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                       ndim, 
                                                                      order, 
                                                                  old_steps, 
                                                                  new_steps, 
                                          reinterpret_cast<const half*>(dy),
                                               reinterpret_cast<half*>(dx));
    CUDA_POST_KERNEL_CHECK;
}

/******************** common.utils ********************/

template <typename T>
__global__ void _OneHot(const int count,
                        const int depth, 
                        const int on_value, 
                        const float* x,
                        float* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int val = x[idx];
        y[idx * depth + val] = on_value;
    }
}


template <> void OneHot<float, CUDAContext>(const int count,
                                            const int depth,
                                            const int on_value,
                                            const float* x,
                                            float* y) {
    _OneHot<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                depth,
                                                             on_value,
                                                                    x,
                                                                   y);
    CUDA_POST_KERNEL_CHECK;
}

/******************** loss.l1_loss ********************/

template <typename T>
__global__ void _AbsGrad(const int count, const T* dy, T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
       const T val = dy[idx];
       //    val > 0: 1 | val == 0: 0 | val < 0: -1
       dx[idx] = (val > T(0)) - (val < T(0));
    }
}

template<> void AbsGrad<float, CUDAContext>(const int count, const float* dy, float* dx) {
    _AbsGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, dy, dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** loss.sigmoid_cross_entropy_loss ********************/

template <typename T>
__global__ void _SigmoidCrossEntropy(const int count, 
                                     const T* x, 
                                     const T* targets,
                                     T* loss) {
    CUDA_KERNEL_LOOP(idx, count) {
        loss[idx] = std::log(1 + std::exp(x[idx] - 2 * x[idx] * (x[idx] >= 0))) 
                       + x[idx] * ((x[idx] >= 0) - targets[idx]);
    }
}

template <> void SigmoidCrossEntropy<float, CUDAContext>(const int count, 
                                                         const float* x, 
                                                         const float* targets, 
                                                         float* loss) {
    _SigmoidCrossEntropy<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                                 x, 
                                                                           targets, 
                                                                             loss);
     CUDA_POST_KERNEL_CHECK;
}

/******************** loss.smooth_l1_loss ********************/

template <typename T>
__global__ void _SmoothL1(const int count, const float sigma2, const T* x, T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const T val = x[idx];
        const T abs_val = abs(val);
        if (abs_val < 1.0 / sigma2) y[idx] = 0.5 * val * val *sigma2;
        else y[idx] = abs_val - 0.5 / sigma2;
    }
}

template<> void SmoothL1<float, CUDAContext>(const int count, 
                                             const float sigma2, 
                                             const float* x, 
                                             float* y) {
    _SmoothL1<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, sigma2, x, y);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _SmoothL1Grad(const int count, const float sigma2, const T* dy, T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const T val = dy[idx];
        const T abs_val = abs(val);
        if (abs_val < 1.0 / sigma2) dx[idx] = val * sigma2;
        //    val > 0: 1 | val == 0: 0 | val < 0: -1
        else dx[idx] = (val > T(0)) - (val < T(0));
    }
}

template<> void SmoothL1Grad<float, CUDAContext>(const int count, 
                                                 const float sigma2, 
                                                 const float* dy, 
                                                 float* dx) {
    _SmoothL1Grad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, sigma2, dy, dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** loss.softmax_cross_entropy_loss ********************/

template <typename T>
__global__ void _SoftmaxCrossEntropy(const int count, 
                                     const T* prob, 
                                     const T* labels, 
                                     T* loss) {
    CUDA_KERNEL_LOOP(idx, count) {
        loss[idx] = - labels[idx] * log(max(prob[idx], FLT_MIN));
    }
}

template <> void SoftmaxCrossEntropy<float, CUDAContext>(const int count, 
                                                         const float* prob, 
                                                         const float* labels, 
                                                         float* loss) {
    _SoftmaxCrossEntropy<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                              prob, 
                                                                            labels, 
                                                                             loss);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _SoftmaxCrossEntropyGrad(const int count, 
                                         const T* prob, 
                                         const T* labels, 
                                         T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        dx[idx] = prob[idx] - (labels[idx] > 0);
    }
}

template <> void SoftmaxCrossEntropyGrad<float, CUDAContext>(const int count, 
                                                             const float* prob, 
                                                             const float* labels, 
                                                             float* dx) {
    _SoftmaxCrossEntropyGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                                  prob, 
                                                                                labels, 
                                                                                   dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** loss.softmax_loss ********************/

template <typename T>
__global__ void _SparseSoftmaxCrossEntropy(const int count, 
                                           const T* prob, 
                                           const T* labels, 
                                           T* loss,
                                           const int classes, 
                                           const int inner_dim, 
                                           const int* ignores, 
                                           const int ignore_num, 
                                           T* valid) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int o_idx = idx / inner_dim;
        const int i_idx = idx % inner_dim;
        const int label = labels[o_idx * inner_dim + i_idx];
        int k;
        for (k = 0; k < ignore_num; k++) {
            if (label == ignores[k]) {
                loss[idx] = valid[idx] = 0;
                break;
            }
        }
        if (k == ignore_num) {
            loss[idx] = -log(max(prob[(o_idx * classes + label) * 
                                        inner_dim + i_idx], FLT_MIN));
            valid[idx] = 1;
        }
    }
}

template <> void SparseSoftmaxCrossEntropy<float, CUDAContext>(const int count, 
                                                               const int classes, 
                                                               const int outer_dim, 
                                                               const int inner_dim,
                                                               const float* prob, 
                                                               const float* labels, 
                                                               float* loss, 
                                                               float* valid, 
                                                               Tensor* ignore) {
    const int* ignores = ignore->count() > 0 ?
                         ignore->data<int, CUDAContext>() : 
                         nullptr;
    const int num_preds = outer_dim * inner_dim;
    _SparseSoftmaxCrossEntropy<float> << <GET_BLOCKS(num_preds), CUDA_NUM_THREADS >> >(num_preds, 
                                                                                            prob, 
                                                                                          labels, 
                                                                                            loss,
                                                                                         classes, 
                                                                                       inner_dim, 
                                                                                         ignores, 
                                                                                 ignore->count(), 
                                                                                          valid);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _SoftmaxLossGrad(const int count, 
                                 const T* prob, 
                                 const T* labels, 
                                 T* dx, 
                                 const int classes, 
                                 const int inner_dim, 
                                 const int* ignores, 
                                 const int ignore_num, 
                                 T* valid) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int o_idx = idx / inner_dim;
        const int i_idx = idx % inner_dim;
        const int label = labels[o_idx * inner_dim + i_idx];
        int k;
        for (k = 0; k < ignore_num; k++) 
                if (label == ignores[k]) break;
        if (k != ignore_num) {
                for (int c = 0; c < classes; c++)
                    dx[(o_idx * classes + c) * inner_dim + i_idx] = 0;
                valid[idx] = 0;
        } else {
                dx[(o_idx * classes + label) * inner_dim + i_idx] -= 1;
                valid[idx] = 1;
        }
    }
}

template<> void SoftmaxLossGrad<float, CUDAContext>(const int count, 
                                                    const int classes, 
                                                    const int outer_dim, 
                                                    const int inner_dim, 
                                                    const float* labels, 
                                                    const float* prob, 
                                                    float* valid, 
                                                    Tensor* ignore, 
                                                    float* dXdata) {
    const int* ignores = ignore->count() > 0 ? 
                         ignore->data <int, CUDAContext >() : 
                         nullptr;
    const int num_preds = outer_dim * inner_dim;
    _SoftmaxLossGrad<float> << <GET_BLOCKS(num_preds), CUDA_NUM_THREADS >> >(num_preds, 
                                                                                  prob, 
                                                                                labels, 
                                                                                dXdata,
                                                                               classes, 
                                                                             inner_dim, 
                                                                               ignores, 
                                                                       ignore->count(), 
                                                                                valid);
    CUDA_POST_KERNEL_CHECK;
}

/******************** recurrent.lstm_uint ********************/

template <typename T>
__global__ void _LSTMUnitAct(const int count, 
                             const int channels, 
                             const int g_offset,
                             const int x_offset, 
                             const T* x,
                             T* x_act) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int ch_4 = idx % x_offset;
        if (ch_4 < g_offset) x_act[idx] = _SigmoidUnit<float>(x[idx]);
        else x_act[idx] = std::tanh(x[idx]);
    }
}

template <typename T>
__global__ void _LSTMUnit(const int count, 
                          const int channels,
                          const int o_offset, 
                          const int g_offset, 
                          const int x_offset,
                          const T* c_1, 
                          T* x_act, 
                          const T* cont, 
                          T* c, 
                          T* h) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int n = idx / channels;
        const int ch = idx % channels;
        T* x_act_  = x_act + n * x_offset;
        const T i = x_act_[ch];
        if (cont != nullptr && cont[n] != T(1)) 
            x_act_[channels + ch] *= cont[n];
        const T f = x_act_[channels + ch];
        const T o = x_act_[o_offset + ch];
        const T g = x_act_[g_offset + ch];
        const T c_ = c[idx] = f * c_1[idx] + i * g;
        h[idx] = o * std::tanh(c_);
    }
}

template <> void LSTMUnit<float, CUDAContext>(const int count, 
                                              const int num, 
                                              const int channels,
                                              const float* c_1, 
                                              const float* x, 
                                              const float* cont,
                                              float* x_act, 
                                              float* c, 
                                              float* h) {
    const int o_offset = 2 * channels, g_offset = 3 * channels;
    const int x_offset = 4 * channels, y_count = count / 4;
    _LSTMUnitAct<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                  channels, 
                                                                  g_offset, 
                                                                  x_offset, 
                                                                         x, 
                                                                    x_act);
    _LSTMUnit<float> << <GET_BLOCKS(y_count), CUDA_NUM_THREADS >> >(y_count, 
                                                                   channels, 
                                                                   o_offset, 
                                                                   g_offset, 
                                                                   x_offset,
                                                                        c_1, 
                                                                      x_act, 
                                                                       cont, 
                                                                          c, 
                                                                         h);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _LSTMUnitGrad(const int count, 
                              const int channels,
                              const int o_offset, 
                              const int g_offset, 
                              const int x_offset,
                              const T* c_1, 
                              const T* x_act, 
                              const T* c, 
                              const T* dc, 
                              const T* dh, 
                              T* dc_1, 
                              T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int n = idx / channels;
        const int ch = idx % channels;
        const T* x_act_ = x_act + n * x_offset;
        T* dx_ = dx + n * x_offset;
        const T i = x_act_[ch];
        const T f = x_act_[channels + ch];
        const T o = x_act_[o_offset + ch];
        const T g = x_act_[g_offset + ch];
        T* p_di = dx_ + ch;
        T* p_df = dx_ + channels + ch;
        T* p_do = dx_ + o_offset + ch;
        T* p_dg = dx_ + g_offset + ch;
        const T tanh_c_t = tanh(c[idx]);
        const T dc_1_sum_term = dh[idx] * o * (1 - tanh_c_t * tanh_c_t) + dc[idx];
        dc_1[idx] = dc_1_sum_term * f;
        *p_di = dc_1_sum_term * g;
        *p_df = dc_1_sum_term * c_1[idx];
        *p_do = dh[idx] * tanh_c_t;
        *p_dg = dc_1_sum_term * i;
    }
}

template <typename T>
__global__ void _LSTMUnitGradAct(const int count, 
                                 const int channels, 
                                 const int g_offset,
                                 const int x_offset, 
                                 const T* x_act, 
                                 T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int ch_4 = idx % x_offset;
        const T x_act_ = x_act[idx];
        if (ch_4 < g_offset) dx[idx] = dx[idx] * x_act_ * (T(1) - x_act_);
        else  dx[idx] = dx[idx] * (T(1) - x_act_ * x_act_);
    }
}

template <> void LSTMUnitGrad<float, CUDAContext>(const int count, 
                                                  const int num, 
                                                  const int channels,
                                                  const float* c_1, 
                                                  const float* x_act,
                                                  const float* c, 
                                                  const float* dc, 
                                                  const float* dh,
                                                  float* dc_1, 
                                                  float* dx) {
    const int o_offset = 2 * channels, g_offset = 3 * channels;
    const int x_offset = 4 * channels, y_count = count / 4;
    _LSTMUnitGrad<float> << <GET_BLOCKS(y_count), CUDA_NUM_THREADS >> >(y_count, 
                                                                       channels, 
                                                                       o_offset, 
                                                                       g_offset, 
                                                                       x_offset,
                                                                            c_1, 
                                                                          x_act, 
                                                                              c, 
                                                                             dc, 
                                                                             dh, 
                                                                           dc_1, 
                                                                            dx);
    _LSTMUnitGradAct<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                      channels, 
                                                                      g_offset,
                                                                      x_offset, 
                                                                         x_act, 
                                                                           dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** update.adam_update ********************/

template <typename T>
__global__ void _AdamUpdate(const int n, 
                            T* g, 
                            T* m, 
                            T* v,
                            const T beta1, 
                            const T beta2, 
                            const T eps, 
                            const T lr) {
    CUDA_KERNEL_LOOP(i, n) {
        T gi = g[i];
        T mi = m[i] = m[i] * beta1 + gi * (1 - beta1);
        T vi = v[i] = v[i] * beta2 + gi * gi * (1 - beta2);
        g[i] = lr * mi / (sqrt(vi) + eps);
    }
}

template <> void AdamUpdate<float, CUDAContext>(Tensor* x, 
                                                Tensor* m, 
                                                Tensor* v, 
                                                Tensor* t,
                                                const float beta1, 
                                                const float beta2, 
                                                const float eps, 
                                                const float lr) {
    TIndex count = x->count();
    auto* Xdata = x->mutable_data<float, CUDAContext>();
    auto* Mdata = m->mutable_data<float, CUDAContext>();
    auto* Vdata = v->mutable_data<float, CUDAContext>();
    _AdamUpdate<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                    Xdata, 
                                                                    Mdata, 
                                                                    Vdata, 
                                                                    beta1, 
                                                                    beta2, 
                                                                      eps, 
                                                                      lr);
    CUDA_POST_KERNEL_CHECK;
}

/******************** update.nesterov_update ********************/

template <typename T>
__global__ void _NesterovUpdate(const int n, 
                               T* g, 
                               T* h,
                               const T momentum,
                               const T lr) {
    CUDA_KERNEL_LOOP(i, n) {
        T hi = h[i];
        T hi_new = h[i] = momentum * hi + lr * g[i];
        g[i] = (1 + momentum) * hi_new - momentum * hi;
    }
}
template <> void NesterovUpdate<float, CUDAContext>(const int count,
                                                    float* x,
                                                    float* h,
                                                    Tensor* t,
                                                    const float momentum,
                                                    const float lr,
                                                    CUDAContext* ctx) {
    _NesterovUpdate<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                            x, 
                                                                            h, 
                                                                     momentum,
                                                                          lr);
    CUDA_POST_KERNEL_CHECK;
}

/******************** update.rmsprop_update ********************/

template <typename T>
__global__ void _RMSPropUpdate(const int n, 
                               T* g, 
                               T* h,
                               const T decay, 
                               const T eps, 
                               const T lr) {
    CUDA_KERNEL_LOOP(i, n) {
        T gi = g[i];
        T hi = h[i] = decay * h[i] + (1 - decay) * gi * gi;
        g[i] = lr * g[i] / (sqrt(hi) + eps);
    }
}

template <> void RMSPropUpdate<float, CUDAContext>(const int count,
                                                   float* x, 
                                                   float* h,
                                                   Tensor* t,
                                                   const float decay, 
                                                   const float eps, 
                                                   const float lr) {
    _RMSPropUpdate<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                           x, 
                                                                           h, 
                                                                       decay, 
                                                                         eps, 
                                                                         lr);
    CUDA_POST_KERNEL_CHECK;
}

/******************** utils.cast ********************/

template <typename T>
__global__ void _FloatToHalfKernel(const int count, const float* x, half* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        y[idx] = __float2half(x[idx]);
    }
}

template <> void Float2Half<float, CUDAContext>(const int count, 
                                                const float* x, 
                                                float16* y) {
    _FloatToHalfKernel<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                               x, 
                                                     reinterpret_cast<half*>(y));
     CUDA_POST_KERNEL_CHECK;
}

/******************** utils.compare ********************/

template <typename T>
__global__ void _Equal(const int count, const T* a, const T* b, T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        y[idx] = fabs(a[idx] - b[idx]) < FLT_EPSILON ? 1.0 : 0.0;
    }
}

template <> void Equal<float, CUDAContext>(const int count, 
                                           const float* a,
                                           const float* b, 
                                           float* y) {
    _Equal<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, a, b, y);
     CUDA_POST_KERNEL_CHECK;
}

/******************** utils.memory_data ********************/

template <typename Tx, typename Ty>
__global__ void _MemoryData(const int count, 
                            const int num, 
                            const int channels, 
                            const int height, 
                            const int width, 
                            const Tx* x, 
                            Ty* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int w = idx % width;
        const int h = (idx / width) % height;
        const int c = (idx / width / height) % channels;
        const int n = idx / width / height / channels;
        const int x_idx = ((n * height + h) * width + w) * channels + c;
        if (c == 0) y[idx] = x[x_idx] - 102.9801;
        else if (c == 1) y[idx] = x[x_idx] - 115.9465;
        else y[idx] = x[x_idx] - 122.7717;
    }
}

template <typename Tx, typename Ty>
__global__ void _MemoryDataHalf(const int count, 
                                const int num, 
                                const int channels, 
                                const int height, 
                                const int width, 
                                const Tx* x, 
                                Ty* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int w = idx % width;
        const int h = (idx / width) % height;
        const int c = (idx / width / height) % channels;
        const int n = idx / width / height / channels;
        const int x_idx = ((n * height + h) * width + w) * channels + c;
        if (c == 0) y[idx] = __float2half(x[x_idx] - 102.9801);
        else if (c == 1) y[idx] = __float2half(x[x_idx] - 115.9465);
        else y[idx] = __float2half(x[x_idx] - 122.7717);
    }
}

template <> void MemoryData<float, float, CUDAContext>(const int count, 
                                                       const int num, 
                                                       const int channels, 
                                                       const int height, 
                                                       const int width, 
                                                       const float* x, 
                                                       float* y) {
    _MemoryData<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                      num, 
                                                                 channels, 
                                                                   height, 
                                                                    width, 
                                                                        x, 
                                                                       y);
    CUDA_POST_KERNEL_CHECK;
}

template <> void MemoryData<uint8_t, float, CUDAContext>(const int count, 
                                                       const int num, 
                                                       const int channels, 
                                                       const int height, 
                                                       const int width, 
                                                       const uint8_t* x, 
                                                       float* y) {
    _MemoryData<uint8_t, float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                               num, 
                                                                          channels, 
                                                                            height, 
                                                                             width, 
                                                                                 x, 
                                                                                y);
    CUDA_POST_KERNEL_CHECK;
}

template <> void MemoryData<float, float16, CUDAContext>(const int count, 
                                                         const int num, 
                                                         const int channels, 
                                                         const int height, 
                                                         const int width, 
                                                         const float* x, 
                                                         float16* y) {
    _MemoryDataHalf<float, half> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                                num, 
                                                                           channels, 
                                                                             height, 
                                                                              width, 
                                                                                  x, 
                                                        reinterpret_cast<half*>(y));
    CUDA_POST_KERNEL_CHECK;
}

template <> void MemoryData<uint8_t, float16, CUDAContext>(const int count, 
                                                           const int num, 
                                                           const int channels, 
                                                           const int height, 
                                                           const int width, 
                                                           const uint8_t* x, 
                                                           float16* y) {
    _MemoryDataHalf<uint8_t, half> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                                  num, 
                                                                             channels, 
                                                                               height, 
                                                                                width, 
                                                                                    x, 
                                                          reinterpret_cast<half*>(y));
    CUDA_POST_KERNEL_CHECK;
}

/******************** vision.conv ********************/

template<typename T>
__global__ void _Im2Col(const int count, 
                        const int height, const int width,
                        const int kernel_h, const int kernel_w, 
                        const int stride_h, const int stride_w, 
                        const int pad_h, const int pad_w,
                        const int dilation_h, const int dilation_w,
                        const int col_h, const int col_w, 
                        const T* im,
                        T* col) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int h_idx = idx / col_w;
        const int im_c = h_idx / col_h;
        const int h = h_idx % col_h;
        const int w = idx % col_w;
        const int c = im_c * kernel_h * kernel_w;
        const int im_h_off = h * stride_h - pad_h;
        const int im_w_off = w * stride_w - pad_w;

        //  compute the first col pos of a roll convolution
        T* col_ptr = col;
        col_ptr += ((c * col_h + h) * col_w + w);

        //  compute the first im pos of a roll convolution
        const T* im_ptr = im;
        im_ptr += ((im_c * height + im_h_off) * width + im_w_off);

        for (int i = 0; i < kernel_h; ++i) {
            for (int j = 0; j < kernel_w; ++j) {
                //  compute the current im pos
                int im_h = i * dilation_h + im_h_off;
                int im_w = j * dilation_w + im_w_off;
                *col_ptr = (im_h >= 0 && im_w >= 0 && im_h < height && im_w < width) ?
                           im_ptr[i * dilation_h * width + j * dilation_w] : 0;
                col_ptr += (col_h * col_w);
            }
        }
    }
}

template <> void Im2Col<float, CUDAContext>(const int channels, 
                                            const int height, const int width,
                                            const int kernel_h, const int kernel_w, 
                                            const int stride_h, const int stride_w, 
                                            const int pad_h, const int pad_w,
                                            const int dilation_h, const int dilation_w, 
                                            const float* im,
                                            float* col) {
    const int col_h = (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
    const int col_w = (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;
    const int count = (channels * col_h * col_w);
    _Im2Col<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                        height, width, 
                                                   kernel_h, kernel_w, 
                                                   stride_h, stride_w, 
                                                         pad_h, pad_w,
                                               dilation_h, dilation_w, 
                                                         col_h, col_w, 
                                                                   im,
                                                                 col);
    CUDA_POST_KERNEL_CHECK;
}

template<typename T>
__global__ void _Col2Im(const int count, 
                        const int height, const int width,
                        const int kernel_h, const int kernel_w, 
                        const int stride_h, const int stride_w, 
                        const int pad_h, const int pad_w,
                        const int dilation_h, const int dilation_w,
                        const int col_h, const int col_w, 
                        const T* col,
                        T* im) {
    CUDA_KERNEL_LOOP(idx, count) {
        T val = 0;
        const int im_w = idx % width + pad_w;
        const int im_h = (idx / width) % height + pad_h;
        const int im_c = idx / (width * height);
        const int ex_kernel_h = (kernel_h - 1) * dilation_h + 1;
        const int ex_kernel_w = (kernel_w - 1) * dilation_w + 1;
        const int w_start = (im_w < ex_kernel_w) ? 0 : (im_w - ex_kernel_w) / stride_w + 1;

        //    redundant pixels will be ignored when conv
        //    note to clip them by min(x,col_w)
        const int w_end = min(im_w / stride_w + 1, col_w);
        const int h_start = (im_h < ex_kernel_h) ? 0 : (im_h - ex_kernel_h) / stride_h + 1;
        const int h_end = min(im_h / stride_h + 1, col_h);

        for (int h = h_start; h < h_end; ++h) {
            for (int w = w_start; w < w_end; ++w) {
                int kh_off = (im_h - h * stride_h);
                int kw_off = (im_w - w * stride_w);
                //    only the serval im pixels used in dilated-conv
                //    ignore the corresponding col pixels
                if (kh_off % dilation_h == 0 && kw_off % dilation_w == 0) {
                    kh_off /= dilation_h;
                    kw_off /= dilation_w;
                    int c = (im_c * kernel_h + kh_off) * kernel_w + kw_off;
                    val += col[(c * col_h + h) * col_w + w];
                }
            }
        }
        im[idx] = val;
    }
}

template <> void Col2Im<float, CUDAContext>(const int channels, 
                                            const int height, const int width,
                                            const int kernel_h, const int kernel_w, 
                                            const int stride_h, const int stride_w, 
                                            const int pad_h, const int pad_w,
                                            const int dilation_h, const int dilation_w, 
                                            const float* col,
                                            float* im) {
    const int col_h = (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
    const int col_w = (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;
    const int count = (channels * height * width);
    _Col2Im<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                        height, width, 
                                                   kernel_h, kernel_w, 
                                                   stride_h, stride_w,
                                                         pad_h, pad_w,
                                               dilation_h, dilation_w, 
                                                         col_h, col_w,
                                                                  col,
                                                                  im);
    CUDA_POST_KERNEL_CHECK;
}

/******************** vision.nn_resize ********************/

template <typename T>
__global__ void _NNResize(const int count, 
                          const float h_scale, 
                          const float w_scale,
                          const int num, const int channels, 
                          const int h_in, const int w_in, 
                          const int h_out, const int w_out, 
                          const T* x, 
                          T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int w = idx % w_out;
        const int h = (idx / w_out) % h_out;
        const int in_h = min(int(floorf(h * h_scale)), h_in - 1);
        const int in_w = min(int(floorf(w * w_scale)), w_in - 1);
        const int c = (idx / w_out / h_out) % channels;
        const int n = idx / w_out / h_out / channels;
        const int x_idx = ((n * channels + c) * h_in + in_h) * w_in + in_w;
        y[idx] = x[x_idx];
    }
}

template <> void NNResize<float, CUDAContext>(const int count, 
                                              const int num, const int channels,
                                              const int h_in, const int w_in, 
                                              const int h_out, const int w_out,
                                              const float* x, float* y) {
    const float h_scale = (float)h_in / h_out;
    const float w_scale = (float)w_in / w_out;
    _NNResize<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                h_scale, 
                                                                w_scale, 
                                                          num, channels, 
                                                             h_in, w_in, 
                                                           h_out, w_out, 
                                                                      x, 
                                                                     y);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
 __global__ void _NNResizeGrad(const int count, 
                               const float h_scale, const float w_scale,
                               const int num, const int channels, 
                               const int h_in, const int w_in,
                               const int h_out, const int w_out, 
                               const T* dy, 
                               T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int w = idx % w_in;
        const int h = (idx / w_in) % h_in;
        const int out_h = min(int(floorf(h * h_scale)), h_out - 1);
        const int out_w = min(int(floorf(w * w_scale)), w_out - 1);
        const int c = (idx / w_in / h_in) % channels;
        const int n = idx / w_in / h_in / channels;
        const int x_idx = ((n * channels + c) * h_out + out_h) * w_out + out_w;
        atomicAdd(dx + x_idx, dy[idx]);
    }
}

template <> void NNResizeGrad<float, CUDAContext>(const int count,
                                                  const int num, 
                                                  const int channels,
                                                  const int h_in, const int w_in, 
                                                  const int h_out, const int w_out,
                                                  const float* dy, float* dx) {
    const float h_scale = (float)h_out / h_in;
    const float w_scale = (float)w_out / w_in;
    _NNResizeGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                    h_scale, 
                                                                    w_scale, 
                                                              num, channels, 
                                                                 h_in, w_in, 
                                                               h_out, w_out, 
                                                                         dy, 
                                                                        dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** vision.pooling ********************/

template<typename T>
__global__ void _MAXPooling(const int count, 
                            const int num, const int channels,
                            const int height, const int width, 
                            const int pool_height, const int pool_width,
                            const int kernel_h, const int kernel_w, 
                            const int stride_h, const int stride_w, 
                            const int pad_h, const int pad_w, 
                            const T* x,
                            int* mask,
                            T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int pw = idx % pool_width;
        const int ph = (idx / pool_width) % pool_height;
        const int pc = (idx / pool_width / pool_height) % channels;
        const int pn = (idx / pool_width / pool_height / channels);

        int start_h = ph * stride_h - pad_h;
        int start_w = pw * stride_w - pad_w;
        const int end_h = min(start_h + kernel_h, height);
        const int end_w = min(start_w + kernel_w, width);

        start_h = max(start_h, 0);
        start_w = max(start_w, 0);

        T max_val = -FLT_MAX;
        int max_idx = -1;
        const T* x_ptr = x + (pn * channels + pc) * height * width;

        for (int h = start_h; h < end_h; ++h) {
            for (int w = start_w; w < end_w; ++w) {
                if (x_ptr[h * width + w] > max_val) {
                    max_idx = h * width + w;
                    max_val = x_ptr[max_idx];
                }
            }
        }
        y[idx] = max_val;
        mask[idx] = max_idx;
    }
}

template<> void MAXPooling<float, CUDAContext>(const int count, 
                                               const int num, const int channels,
                                               const int height, const int width, 
                                               const int pool_height, const int pool_width,
                                               const int kernel_h, const int kernel_w, 
                                               const int stride_h, const int stride_w, 
                                               const int pad_h, const int pad_w,
                                               const float* x, 
                                               int* mask, 
                                               float* y) {
    _MAXPooling<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                             num, channels, height, width, 
                                                  pool_height, pool_width, 
                                                       kernel_h, kernel_w, 
                                                       stride_h, stride_w, 
                                                             pad_h, pad_w, 
                                                                        x,
                                                                     mask,
                                                                       y); 

    CUDA_POST_KERNEL_CHECK;
}

template<typename T>
__global__ void _AVEPooling(const int count, 
                            const int num, const int channels,
                            const int height, const int width, 
                            const int pool_height, const int pool_width,
                            const int kernel_h, const int kernel_w, 
                            const int stride_h, const int stride_w, 
                            const int pad_h, const int pad_w, 
                            const T* x,
                            T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int pw = idx % pool_width;
        const int ph = (idx / pool_width) % pool_height;
        const int pc = (idx / pool_width / pool_height) % channels;
        const int pn = (idx / pool_width / pool_height / channels);

        int start_h = ph * stride_h - pad_h;
        int start_w = pw * stride_w - pad_w;
        int end_h = min(start_h + kernel_h, height + pad_h);
        int end_w = min(start_w + kernel_w, width + pad_w);

        start_h = max(start_h, 0);
        start_w = max(start_w, 0);
        end_h = min(end_h, height);
        end_w = min(end_w, width);

        const T* x_ptr = x + (pn * channels + pc) * height * width;
        const int pooling_size = (end_h - start_h) * (end_w - start_w);
        T avg_val = 0;

        for (int h = start_h; h < end_h; ++h) {
            for (int w = start_w; w < end_w; ++w) {
                avg_val += x_ptr[h * width + w];
            }
        }
        y[idx] = avg_val / pooling_size;
    }
}

template<> void AVEPooling<float, CUDAContext>(const int count, 
                                               const int num, const int channels,
                                               const int height, const int width, 
                                               const int pool_height, const int pool_width,
                                               const int kernel_h, const int kernel_w, 
                                               const int stride_h, const int stride_w, 
                                               const int pad_h, const int pad_w,
                                               const float* x, 
                                               float* y) {
    _AVEPooling<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                             num, channels, height, width, 
                                                  pool_height, pool_width,
                                                       kernel_h, kernel_w, 
                                                       stride_h, stride_w, 
                                                             pad_h, pad_w, 
                                                                        x,
                                                                       y);
    CUDA_POST_KERNEL_CHECK; 
}

template<typename T>
__global__ void _MAXPoolingGrad(const int count, 
                                const int num, const int channels,
                                const int height, const int width, 
                                const int pool_height, const int pool_width,
                                const int kernel_h, const int kernel_w, 
                                const int stride_h, const int stride_w,
                                const int pad_h, const int pad_w, 
                                const T* dy,
                                const int* mask,
                                T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int w = idx % width;
        const int h = (idx / width) % height;
        const int c = (idx / width / height) % channels;
        const int n = idx / width / height / channels;

        //    allow overlapping
        const int start_ph = (h + pad_h < kernel_h) ? 0 : (h + pad_h - kernel_h) / stride_h + 1;
        const int start_pw = (w + pad_w < kernel_w) ? 0 : (w + pad_w - kernel_w) / stride_w + 1;

        //    allow clip
        const int end_ph = min((h + pad_h) / stride_h + 1, pool_height);
        const int end_pw = min((w + pad_w) / stride_w + 1, pool_width);

        T diff = 0;
        const int offset = (n * channels + c) * pool_height * pool_width;
        const T* y_ptr = dy + offset;
        const int* mask_ptr = mask + offset;

        for (int ph = start_ph; ph < end_ph; ++ph) {
            for (int pw = start_pw; pw < end_pw; ++pw) {
                if (mask_ptr[ph * pool_width + pw] == (h * width + w)) {
                    diff += y_ptr[ph * pool_width + pw];
                }
            }
        }
        dx[idx] = diff;
    }
}

template<> void MAXPoolingGrad<float, CUDAContext>(const int count, 
                                                   const int num, const int channels,
                                                   const int height, const int width, 
                                                   const int pool_height, const int pool_width,
                                                   const int kernel_h, const int kernel_w, 
                                                   const int stride_h, const int stride_w, 
                                                   const int pad_h, const int pad_w,
                                                   const float* dy, 
                                                   const int* mask, 
                                                   float* dx) {
    _MAXPoolingGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                 num, channels, height, width, 
                                                      pool_height, pool_width,
                                                           kernel_h, kernel_w, 
                                                           stride_h, stride_w, 
                                                                 pad_h, pad_w, 
                                                                           dy,
                                                                         mask,
                                                                          dx);
    CUDA_POST_KERNEL_CHECK;
}

template<typename T>
__global__ void _AVEPoolingGrad(const int count, 
                                const int num, const int channels,
                                const int height, const int width, 
                                const int pool_height, const int pool_width,
                                const int kernel_h, const int kernel_w, 
                                const int stride_h, const int stride_w,
                                const int pad_h, const int pad_w, 
                                const T* dy,
                                T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int w = idx % width;
        const int h = (idx / width) % height;
        const int c = (idx / width / height) % channels;
        const int n = idx / width / height / channels;

        const int start_ph = (h + pad_h < kernel_h) ? 0 : (h + pad_h - kernel_h) / stride_h + 1;
        const int start_pw = (w + pad_w<kernel_w) ? 0 : (w + pad_w - kernel_w) / stride_w + 1;
        const int end_ph = min(h / stride_h + 1, pool_height);
        const int end_pw = min(w / stride_w + 1, pool_width);

        T diff = 0;
        const T* y_ptr = dy + (n * channels + c) * pool_height * pool_width;

        for (int ph = start_ph; ph < end_ph; ++ph) {
            for (int pw = start_pw; pw < end_pw; ++pw) {
                int start_h = ph * stride_h - pad_h;
                int start_w = pw * stride_w - pad_w;
                int end_h = min(start_h + kernel_h, height + pad_h);
                int end_w = min(start_w + kernel_w, width + pad_w);
                int pooling_size = (end_h - start_h) * (end_w - start_w);
                diff += (y_ptr[ph * pool_width + pw] / pooling_size);
            }
        }
        dx[idx] = diff;
    }
}

template<> void AVEPoolingGrad<float, CUDAContext>(const int count, 
                                                   const int num, const int channels,
                                                   const int height, const int width, 
                                                   const int pool_height, const int pool_width,
                                                   const int kernel_h, const int kernel_w, 
                                                   const int stride_h, const int stride_w, 
                                                   const int pad_h, const int pad_w,
                                                   const float* dy,
                                                   float* dx) {
    _AVEPoolingGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                 num, channels, height, width, 
                                                      pool_height, pool_width,
                                                           kernel_h, kernel_w, 
                                                           stride_h, stride_w, 
                                                                 pad_h, pad_w, 
                                                                           dy,
                                                                          dx);
    CUDA_POST_KERNEL_CHECK;
}


/******************** vision.roi_pooling ********************/

template <typename T>
__global__ void _ROIPooling(const int count, 
                            const T spatial_scale, 
                            const int channels, 
                            const int height, const int width,
                            const int pool_h, const int pool_w, 
                            const T* x,
                            const T* roi,
                            int* mask,
                            T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        int pw = idx % pool_w;
        int ph = (idx / pool_w) % pool_h;
        int c = (idx / pool_w / pool_h) % channels;
        int n = idx / pool_w / pool_h / channels;

        roi += n * 5;
        int im_idx = roi[0];

        int x1 = round(roi[1] * spatial_scale);
        int y1 = round(roi[2] * spatial_scale);
        int x2 = round(roi[3] * spatial_scale);
        int y2 = round(roi[4] * spatial_scale);

        int roi_height = max(y2 - y1 + 1, 1);
        int roi_width = max(x2 - x1 + 1, 1);

        const float bin_size_h = (float)roi_height / (float)pool_h;
        const float bin_size_w = (float)roi_width / (float)pool_w;

        int start_h = floor(bin_size_h * ph);
        int start_w = floor(bin_size_w * pw);
        int end_h = ceil(bin_size_h * (ph + 1));
        int end_w = ceil(bin_size_w * (pw + 1));

        start_h = min(max(start_h + y1, 0), height);
        start_w = min(max(start_w + x1, 0), width);
        end_h = min(max(end_h + y1, 0), height);
        end_w = min(max(end_w + x1, 0), width);

        bool is_empty = (end_h <= start_h) || (end_w <= start_w);
        float max_val = is_empty ? 0 : -FLT_MAX;
        int max_idx = -1;
        x += ((im_idx * channels + c) * height * width);

        for (int h = start_h; h < end_h; ++h) {
            for (int w = start_w; w < end_w; ++w) {
                const int x_idx = h * width + w;
                if (x[x_idx] > max_val) {
                    max_val = x[x_idx];
                    max_idx = x_idx;
                }
            }    //end w
        }    // end h

        y[idx] = max_val;
        mask[idx] = max_idx;
    }
}

template<> void ROIPooling<float, CUDAContext>(const float spatial_scale, 
                                               const int pool_h, const int pool_w,
                                               Tensor* x,
                                               Tensor* roi,
                                               Tensor* mask,
                                               Tensor* y) {
    auto* Xdata = x->data<float, CUDAContext>();
    auto* Rdata = roi->data<float, CUDAContext>();
    auto* Ydata = y->mutable_data<float, CUDAContext>();
    auto* Mdata = mask->mutable_data<int, CUDAContext>();
    TIndex channels = x->dim(1), count = y->count();
    TIndex height = x->dim(2), width = x->dim(3);
    _ROIPooling<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                            spatial_scale, 
                                                                 channels, 
                                                            height, width,
                                                           pool_h, pool_w,
                                                                    Xdata,
                                                                    Rdata,
                                                                    Mdata,
                                                                   Ydata);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _ROIPoolingGrad(const int count, 
                                const int num_rois, 
                                const T spatial_scale, 
                                const int channels, 
                                const int height, const int width,
                                const int pool_h, const int pool_w, 
                                const T* dy,
                                const T* roi,
                                const int* mask,
                                T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        int w = idx % width;
        int h = (idx / width) % height;
        int c = (idx / width / height) % channels;
        int im_idx = idx / width / height / channels;

        T diff = 0;

        for (int n = 0; n < num_rois; ++n) {
            const T* cur_roi = roi + n * 5;
            const int im_idx_spec = cur_roi[0];

            //    ignore wrong im_batch_idx
            if (im_idx != im_idx_spec) continue;

            int x1 = round(cur_roi[1] * spatial_scale);
            int y1 = round(cur_roi[2] * spatial_scale);
            int x2 = round(cur_roi[3] * spatial_scale);
            int y2 = round(cur_roi[4] * spatial_scale);

            const bool is_in = (w >= x1 && w <= x2 && h >= y1 && h <= y2);

            if (!is_in) continue;

            int roi_height = max(y2 - y1 + 1, 1);
            int roi_width = max(x2 - x1 + 1, 1);

            const float bin_size_h = (float)roi_height / (float)pool_h;
            const float bin_size_w = (float)roi_width / (float)pool_w;

            int start_ph = floor((h - y1) / bin_size_h);
            int start_pw = floor((w - x1) / bin_size_w);
            int end_ph = ceil((h + 1 - y1) / bin_size_h);
            int end_pw = ceil((w + 1 - x1) / bin_size_w);

            start_ph = min(max(start_ph, 0), pool_h);
            start_pw = min(max(start_pw, 0), pool_w);
            end_ph = min(max(end_ph, 0), pool_h);
            end_pw = min(max(end_pw, 0), pool_w);

            int y_offset = (n * channels + c) * pool_h * pool_w;
            const T* dy_off = dy + y_offset;
            const int* mask_off = mask + y_offset;

            for (int ph = start_ph; ph < end_ph; ++ph) {
                for (int pw = start_pw; pw < end_pw; ++pw) {
                    int pool_idx = ph * pool_w + pw;
                    if (mask_off[pool_idx] == (h * width + w)) {
                        diff += dy_off[pool_idx];
                    }
                }    //    end pw
            }    // end ph
        }    //    end n
        dx[idx] = diff;
    }
}

template<> void ROIPoolingGrad<float, CUDAContext>(const float spatial_scale, 
                                                   const int pool_h, const int pool_w,
                                                   Tensor* dy,
                                                   Tensor* roi,
                                                   Tensor* mask,
                                                   Tensor* dx) {
    auto* dYdata = dy->data<float, CUDAContext>();
    auto* Rdata = roi->data<float, CUDAContext>();
    auto* Mdata = mask->data<int, CUDAContext>();
    auto* dXdata = dx->mutable_data<float, CUDAContext>();
    TIndex channels = dx->dim(1), count = dx->count();
    TIndex height = dx->dim(2), width = dx->dim(3);
    _ROIPoolingGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                  roi->dim(0), 
                                                                spatial_scale, 
                                                                     channels, 
                                                                height, width,
                                                               pool_h, pool_w,
                                                                       dYdata,
                                                                        Rdata,
                                                                        Mdata,
                                                                      dXdata);
    CUDA_POST_KERNEL_CHECK;
}

/******************** vision.roi_align ********************/

template <typename T>
__global__ void _ROIAlign(const int count, 
                          const float spatial_scale, 
                          const int channels, 
                          const int height, const int width,
                          const int pool_h, const int pool_w, 
                          const T* x,
                          const T* roi,
                          T* mask_h,
                          T* mask_w,
                          T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        int pw = idx % pool_w;
        int ph = (idx / pool_w) % pool_h;
        int c = (idx / pool_w / pool_h) % channels;
        int n = idx / pool_w / pool_h / channels;

        roi += n * 5;
        int im_idx = roi[0];

        T x1 = roi[1] * spatial_scale;
        T y1 = roi[2] * spatial_scale;
        T x2 = roi[3] * spatial_scale;
        T y2 = roi[4] * spatial_scale;

        T roi_height = max(y2 - y1, T(1));
        T roi_width = max(x2 - x1, T(1));

        const T bin_size_h = roi_height / pool_h;
        const T bin_size_w = roi_width / pool_w;

        T start_h = bin_size_h * ph;
        T start_w = bin_size_w * pw;
        T end_h = bin_size_h * (ph + 1);
        T end_w = bin_size_w * (pw + 1);

        start_h = max(start_h + y1, T(0));
        start_w = max(start_w + x1, T(0));
        end_h = max(end_h + y1, T(0));
        end_w = max(end_w + x1, T(0));

        start_h = min(start_h, T(height));
        start_w = min(start_w, T(width));
        end_h = min(end_h, T(height));
        end_w = min(end_w, T(width));

        bool is_empty = (end_h <= start_h) || (end_w <= start_w);
        T max_val = is_empty ? 0 : -FLT_MAX;
        T max_h = -1, max_w = -1;
        x += ((im_idx * channels + c) * height * width);

        for (T h = start_h; h < end_h; ++h) {
            for (T w = start_w; w < end_w; ++w) {
                if (int(ceil(h)) == height) h = height - 1;
                if (int(ceil(w)) == width) w = width - 1;

                int h1 = h, h2 = int(ceil(h));
                int w1 = int(w), w2 = int(ceil(w));

                T q11 = x[h1 * width + w1];
                T q21 = x[h2 * width + w1];
                T q12 = x[h1 * width + w2];
                T q22 = x[h2 * width + w2];

                T val;

                if (h1 == h2) {
                    if (w1 == w2) val = q11;
                    else val = q11 * (w2 - w) + q12 * (w - w1);
                } else if (w1 == w2) {
                    val = q11 * (h2 - h) + q21 * (h - h1);
                } else {
                    val = q11 * (h2 - h) * (w2 - w) +
                    q12 * (h2 - h) * (w - w1) +
                    q21 * (h - h1) * (w2 - w) +
                    q22 * (h - h1) * (w - w1);
                }

                if (val > max_val) {
                    max_val = val;
                    max_h = h;
                    max_w = w;
                }
            }    //end w
        }    // end h
        y[idx] = max_val;
        mask_h[idx] = max_h;
        mask_w[idx] = max_w;
    }
}

template<> void ROIAlign<float, CUDAContext>(const float spatial_scale, 
                                             const int pool_h, const int pool_w,
                                             Tensor* x,
                                             Tensor* roi,
                                             Tensor* mask_h, Tensor* mask_w,
                                             Tensor* y) {
    auto* Xdata = x->data<float, CUDAContext>();
    auto* Rdata = roi->data<float, CUDAContext>();
    auto* Ydata = y->mutable_data<float, CUDAContext>();
    auto* MHdata = mask_h->mutable_data<float, CUDAContext>();
    auto* MWdata = mask_w->mutable_data<float, CUDAContext>();
    TIndex channels = x->dim(1), count = y->count();
    TIndex height = x->dim(2), width = x->dim(3);
    _ROIAlign<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                          spatial_scale, 
                                                               channels, 
                                                          height, width,
                                                         pool_h, pool_w,
                                                                  Xdata,
                                                                  Rdata,
                                                         MHdata, MWdata,
                                                                 Ydata);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _ROIAlignGrad(const int count, 
                              const int num_rois, 
                              const T spatial_scale, 
                              const int channels, 
                              const int height, const int width,
                              const int pool_h, const int pool_w, 
                              const T* dy,
                              const T* roi,
                              const T* mask_h, const T* mask_w,
                              T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        int w = idx % width;
        int h = (idx / width) % height;
        int c = (idx / width / height) % channels;
        int im_idx = idx / width / height / channels;

        T diff = 0;

        for (int n = 0; n < num_rois; n++) {
            const T* cur_roi = roi + n * 5;
            const int im_idx_spec = cur_roi[0];

            //    ignore wrong im_batch_idx
            if (im_idx != im_idx_spec) continue;

            T x1 = cur_roi[1] * spatial_scale;
            T y1 = cur_roi[2] * spatial_scale;
            T x2 = cur_roi[3] * spatial_scale;
            T y2 = cur_roi[4] * spatial_scale;

            const bool is_in = (w + 1 > x1 && w < x2 + 1 && h + 1 > y1 && h < y2 + 1);
            if (!is_in) continue;

            T roi_height = max(y2 - y1, T(1));
            T roi_width = max(x2 - x1, T(1));

            const T bin_size_h = roi_height / pool_h;
            const T bin_size_w = roi_width / pool_w;

            int start_ph = ceil((h - 1 - y1) / bin_size_h - 1);
            int end_ph = ceil((h + 1 - y1) / bin_size_h);
            int start_pw = ceil((w - 1 - x1) / bin_size_w - 1);
            int end_pw = ceil((w + 1 - x1) / bin_size_w);

            start_ph = min(max(start_ph, 0), pool_h);
            start_pw = min(max(start_pw, 0), pool_w);
            end_ph = min(max(end_ph, 0), pool_h);
            end_pw = min(max(end_pw, 0), pool_w);

            int y_offset = (n * channels + c) * pool_h * pool_w;
            const T* dy_off = dy + y_offset;
            const T* mask_h_off = mask_h + y_offset;
            const T* mask_w_off = mask_w + y_offset;

            for (int ph = start_ph; ph < end_ph; ++ph) {
                for (int pw = start_pw; pw < end_pw; ++pw) {
                    T mh = mask_h_off[ph * pool_w + pw];
                    T mw = mask_w_off[ph * pool_w + pw];
                    int h1 = int(mh), h2 = int(ceil(mh));
                    int w1 = int(mw), w2 = int(ceil(mw));
                    if (h1 <= h && h <= h2 && w1 <= w && w <= w2) {
                        T gradient_factor = 1.0;
                        if (h == h1) gradient_factor *= h2 - mh;
                        else gradient_factor *= mh - h1;
                        if (w == w1) gradient_factor *= w2 - mw;
                        else gradient_factor *= mw - w1;
                        diff += dy_off[ph * pool_w + pw] * gradient_factor;
                    }
                }    //    end pw
            }    // end ph
        }    //    end n
        dx[idx] = diff;
    }
}

template<> void ROIAlignGrad<float, CUDAContext>(const float spatial_scale, 
                                                 const int pool_h, const int pool_w,
                                                 Tensor* dy,
                                                 Tensor* roi,
                                                 Tensor* mask_h, Tensor* mask_w,
                                                 Tensor* dx) {
    auto* dYdata = dy->data<float, CUDAContext>();
    auto* Rdata = roi->data<float, CUDAContext>();
    auto* MHdata = mask_h->data<float, CUDAContext>();
    auto* MWdata = mask_w->data<float, CUDAContext>();
    auto* dXdata = dx->mutable_data<float, CUDAContext>();
    TIndex channels = dx->dim(1), count = dx->count();
    TIndex height = dx->dim(2), width = dx->dim(3);
    _ROIAlignGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                roi->dim(0), 
                                                              spatial_scale, 
                                                                   channels, 
                                                              height, width,
                                                             pool_h, pool_w,
                                                                     dYdata,
                                                                      Rdata,
                                                             MHdata, MWdata,
                                                                    dXdata);
    CUDA_POST_KERNEL_CHECK;
}

}    // namespace kernel

}    // namespace dragon

#endif // WITH_CUDA